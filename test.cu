////////////////////////////////////////////////////////////////////////////////
//                             CUDA LATTICE RELAXATION
//                           WRITTEN BY: CLAYTON RAYMENT
//
//    I wrote this program to help teach myself CUDA. While MATLAB and OCTAVE
//    are multithreaded applications, this program runs the majority of the
//    calculation directly on the GPU, which has the capability of being much
//    parallel than a standard CPU multithreaded application. Since I haven't
//    learned how to manage threads very well, the application is currently
//    running close to its memory limit. However, with a better implimentation
//    of thread handling, I would be able to utilize all 1664 CUDA cores on my 
//    GTX 970. Currently I only use blocks, which reduces that significantly.
//    The program is fully scalable, as the user can select how many divisions
//    each centimeter is split up into using the U global variable. Large values
//    of U however, exceed the maximum memory of the card due to my poor thread
//    management.
//
//////////////////////////////////////////////////////////////////////////////////


#include <hip/hip_runtime.h>
#include <iostream>
//===========
//GLOBALS:
//===========

//--------------
// VARIABLES
//--------------
const int U = 10; //Number of blocks per unit. This allows the grid spacing to be user defined.
const int H = 11*U; //Height of the outer box
const int W = 12*U; //Width of the outer box
const int h = 3*U; //Height of the inner box
const int w = 4*U; //Width of the inner box
const int N = H*W; //Number of threads required


//--------------
// FUNCTIONS
//--------------

//Form the base simulation area:
__global__ void fill(double *a){
  //Form the outer rectangle:
  if(blockIdx.x < W || blockIdx.x > (N-W) || blockIdx.x % W == 0 || (blockIdx.x-(W-1))%W == 0){
    a[blockIdx.x] = 9;
  }
  else{
    a[blockIdx.x] = 1;
  }
  //Form the inner rectangle:
  if(blockIdx.x > W*((H-h)/2) && blockIdx.x < W*((H-h)/2+h) && blockIdx.x % W+1 > (W-w)/2 && blockIdx.x % W < (W-w)/2+w){
    a[blockIdx.x] = 0;
  }
}

//Perform one iteration of the matrix relaxation:
__global__ void average(double *a, double *b){
  //if we are at one of the edges, do nothing:
  if(a[blockIdx.x] == 0 || a[blockIdx.x] == 9){
    b[blockIdx.x] = a[blockIdx.x];
  }
  else{
    b[blockIdx.x] = (a[blockIdx.x]+a[blockIdx.x + 1]+a[blockIdx.x - 1]+a[blockIdx.x - W]+a[blockIdx.x + W])/5.0;
  }
}



//-----------------
//HELPER FUNCTIONS
//-----------------

//Quick function to print out the grid:
void print(double *a){
  for(int i = 0; i < N; ++i){
    if(i%W == 0){
      std::cout << '\n';
    }
    std::cout << a[i] << ' ';
  }
}


int main(void){
  double *notPlate_h; //Host copy of the non-plate locations
  double *notPlate_d; //Device copy of the non-plate locations
  int size = N * sizeof(double);

  //The first thing we need to do is fill the initial experiment vector:
  hipMalloc((void **)&notPlate_d, size); //Allocate memory on device for the not-plate vector
  notPlate_h = (double *)malloc(size); //Allocate memory on the host for the not-plate vector
  hipMemcpy(notPlate_d, notPlate_h, size, hipMemcpyHostToDevice);
  fill<<<N,1>>>(notPlate_d);
  hipMemcpy(notPlate_h, notPlate_d, size, hipMemcpyDeviceToHost);
  std::cout << '\n';

  //Now we need to create a secondary vector to hold our rolling values while the GPU works:
  double *tempVal_h;
  double *tempVal_d;
  
  hipMalloc((void **)&tempVal_d, size); //Allocate memory on the device for the temp vector
  tempVal_h = (double *)malloc(size); //allocate memory on the host for the temp vector
  
  
  //Since GPU time is cheap, we don't set a threshold, just run the simulation 1000 times:
  for(int i = 0; i < 1000; ++i){
    hipMemcpy(notPlate_d, notPlate_h, size, hipMemcpyHostToDevice);
    average<<<N,1>>>(notPlate_d, tempVal_d);
    hipMemcpy(tempVal_h, tempVal_d, size, hipMemcpyDeviceToHost);
    notPlate_h = tempVal_h;
  }
  //The output here will be redirected to a plaintext file which will then be opened and plot in OCTAVE
  //print(notPlate_h);

  //When uncommented, this prints to the std output what the values at the user input points are:  
  while(true){
    std::cout << "\nPlease enter an X coordinate: ";
    double x; 
    std::cin >> x;
    x = x*U;
    std::cout << "\nPlease enter a Y coordinate: ";
    double y;
    std::cin >> y;
    y = y*U;
    std::cout << "The point (" << x/U << "," << y/U << ") has value: " << notPlate_h[(int)(y*W+x)] << " Statvolts\n";
  }
  //Free the memory just to be safe:
  free(notPlate_h);
  hipFree(notPlate_d);
  return 0;
}
